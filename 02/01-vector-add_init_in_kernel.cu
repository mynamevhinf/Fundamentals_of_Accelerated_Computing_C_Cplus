
#include <hip/hip_runtime.h>
#include <stdio.h>

/*
 * Host function to initialize vector elements. This function
 * simply initializes each element to equal its index in the
 * vector.
 */

__global__ void initWith(float num, float *a, int N)
{
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  int stride = blockDim.x * gridDim.x;
  
  for(int i = index; i < N; i+=stride)
    a[i] = num;
}

/*
 * Device kernel stores into `result` the sum of each
 * same-indexed value of `a` and `b`.
 */

__global__ void addVectorsInto(float *result, float *a, float *b, int N)
{
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  int stride = blockDim.x * gridDim.x;

  for(int i = index; i < N; i += stride)
    result[i] = a[i] + b[i];
}

/*
 * Host function to confirm values in `vector`. This function
 * assumes all values are the same `target` value.
 */

void checkElementsAre(float target, float *vector, int N)
{
  for(int i = 0; i < N; i++)
  {
    if(vector[i] != target)
    {
      printf("FAIL: vector[%d] - %0.0f does not equal %0.0f\n", i, vector[i], target);
      exit(1);
    }
  }
  printf("Success! All values calculated correctly.\n");
}

int main()
{
  const int N = 2<<24;
  size_t size = N * sizeof(float);

  float *a;
  float *b;
  float *c;

  hipMallocManaged(&a, size);
  hipMallocManaged(&b, size);
  hipMallocManaged(&c, size);
  
  /*
   * nvprof should register performance changes when execution configuration
   * is updated.
   */
  int deviceId;
  hipDeviceProp_t props;  
  hipGetDevice(&deviceId);
  hipGetDeviceProperties(&props, deviceId);
  int multiProcessorCount = props.multiProcessorCount;

  size_t threadsPerBlock = 1024;
  size_t numberOfBlocks = ((N>>10)/multiProcessorCount+1)*multiProcessorCount;
  
  initWith<<<numberOfBlocks, threadsPerBlock>>>(3, a, N);
  initWith<<<numberOfBlocks, threadsPerBlock>>>(4, b, N);
  initWith<<<numberOfBlocks, threadsPerBlock>>>(0, c, N);
  hipDeviceSynchronize();

  addVectorsInto<<<numberOfBlocks, threadsPerBlock>>>(c, a, b, N);
  
  hipError_t addVectorsErr;
  hipError_t asyncErr;
  
  addVectorsErr = hipGetLastError();
  if(addVectorsErr != hipSuccess) 
      printf("Error: %s\n", hipGetErrorString(addVectorsErr));

  asyncErr = hipDeviceSynchronize();
  if(asyncErr != hipSuccess) 
      printf("Error: %s\n", hipGetErrorString(asyncErr));

  checkElementsAre(7, c, N);

  hipFree(a);
  hipFree(b);
  hipFree(c);
}

